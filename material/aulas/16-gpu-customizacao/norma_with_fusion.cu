#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <cmath>
#include <random>
#include <algorithm>
#include <iomanip>


using namespace std;

// obs.: Já existe um transform para elevar ao quadrado (square),
// mas vamos implementar um functor para praticar

/**
// <> TRANSFORM-REDUCE
thrust::transform_reduce( v.begin(), v.end(), unary_op, init, binary_op)
    // unary_op: função unária = SQUARE(X)
    // init: valor inicial 0.0
    // binary_op: função binária = PLUS (X,Y)
    // retorna o resultado da operação binária
/

//<> UNARY OP
struct my_square 
{
    __host__ __device__
    float operator()(const float& x) {
        return x*x;
    }
}; 

// <> BINARY OP -- plus<float>()

float magnitude(thrust::host_vector<float> &v) {
    // copie os vetores para a GPU
    thrust::device_vector<float> d_numbers = v;

    // eleve ao quadrado os elementos do vetor
    thrust::transform(
        d_numbers.begin(), d_numbers.end(),
        d_numbers.begin(), // onde vai armazenar o retorno da função
        my_square()
    );

    float sum = thrust::reduce(
        d_numbers.begin(), d_numbers.end(),
        0.0,
        thrust::plus<float>()
    );

    float norma = sqrt(sum);

    return norma;

}

float magnitude_tr(thrust::host_vector<float> &v) {
    // copie os vetores para a GPU
    thrust::device_vector<float> d_numbers = v;

    float soma_quadratica = thrust::transform_reduce(
        d_numbers.begin(), d_numbers.end(),
        my_square(), // unary op
        0.0,
        thrust::plus<float>() // binary op
    );

    float norma = sqrt(soma_quadratica);

    return norma;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        cerr << argv[0] << ": numero invalido de argumentos\n"; 
        cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }

    // numero de elementos
    int N = atoi(argv[1]); 
    chrono::steady_clock::time_point t1, t2;

    // Geração
    // ---------------------------------------------
    t1 = chrono::steady_clock::now();

    // gere os vetores de tamanho N
    thrust::host_vector<float> h_numbers(N);
    thrust::generate(h_numbers.begin(), h_numbers.end(), rand);
    
    t2 = chrono::steady_clock::now();
    cout << "tempo de geracao: " << chrono::duration_cast<chrono::duration<double> >(t2 - t1).count() << "s" << endl;
    // ---------------------------------------------


    //! Norma c/ transform_reduce
    // ---------------------------------------------
    t1 = chrono::steady_clock::now();

    // float norma = magnitude(h_numbers);
    float norma = magnitude_tr(h_numbers);
    


    t2 = chrono::steady_clock::now();
    cout << "tempo de norma c/ transform_reduce: " << chrono::duration_cast<chrono::duration<double> >(t2 - t1).count() << "s" << endl;
    // ---------------------------------------------

    cout << fixed << setprecision(6);    
    cout << "norma: " << norma << endl;
    

    return 0;

}



