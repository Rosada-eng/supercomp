#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <cmath>
#include <random>
#include <algorithm>
#include <iomanip>


using namespace std;

// obs.: Já existe um transform para elevar ao quadrado (square),
// mas vamos implementar um functor para praticar

struct my_square 
{
    __host__ __device__
    float operator()(const float& x) {
        return x*x;
    }
};

float magnitude(thrust::host_vector<float> &v) {
    // copie os vetores para a GPU
    thrust::device_vector<float> d_numbers = v;

    // eleve ao quadrado os elementos do vetor
    thrust::transform(
        d_numbers.begin(), d_numbers.end(),
        d_numbers.begin(), // onde vai armazenar o retorno da função
        my_square()
    );

    float sum = thrust::reduce(
        d_numbers.begin(), d_numbers.end(),
        0.0,
        thrust::plus<float>()
    );

    float norma = sqrt(sum);

    return norma;

}

int main(int argc, char** argv) {
    if (argc != 2) {
        cerr << argv[0] << ": numero invalido de argumentos\n"; 
        cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }

    // numero de elementos
    int N = atoi(argv[1]); 
    chrono::steady_clock::time_point t1, t2;

    // Geração
    // ---------------------------------------------
    t1 = chrono::steady_clock::now();

    // gere os vetores de tamanho N
    thrust::host_vector<float> h_numbers(N);
    thrust::generate(h_numbers.begin(), h_numbers.end(), rand);
    
    t2 = chrono::steady_clock::now();
    cout << "tempo de geracao: " << chrono::duration_cast<chrono::duration<double> >(t2 - t1).count() << "s" << endl;
    // ---------------------------------------------


    // Norma
    // ---------------------------------------------
    t1 = chrono::steady_clock::now();

    float norma = magnitude(h_numbers);

    t2 = chrono::steady_clock::now();
    cout << "tempo de norma: " << chrono::duration_cast<chrono::duration<double> >(t2 - t1).count() << "s" << endl;
    // ---------------------------------------------

    cout << fixed << setprecision(6);    
    cout << "norma: " << norma << endl;
    

    return 0;

}



