#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main() {
    // cria um vetor no host e armazena os valores das ações

    thrust::host_vector<double> h_vec;
    double price;
    while (std::cin >> price) {
        if (price) {
            h_vec.push_back(price);
        }
    }

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    // cria um vetor no device e copia os valores do vetor do host
    thrust::device_vector<double> d_vec(h_vec);

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    std::cout << "Tempo de copia: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " us" << std::endl;

    //>> REDUCE <<//
    //! Realiza operação em UM VETOR
    double max = thrust::reduce(d_vec.begin(), d_vec.end(), -1.0, thrust::maximum<double>());

    std::cout << "Máximo: " << max << std::endl;

    //** Exercícios **//

    //* Preço médio
    double total_sum = thrust::reduce(d_vec.begin(), d_vec.end(), 0.0, thrust::plus<double>());

    int n = d_vec.size();

    double avg = total_sum / (double)n;

    std::cout << "Preço médio: " << avg << std::endl;

    //* Preço médio no último ano (últimos 365 dias)
    double total_sum_last_year = thrust::reduce(d_vec.end() - 365, d_vec.end(), 0.0, thrust::plus<double>());

    double avg_last_year = total_sum_last_year / 365.0;

    std::cout << "Preço médio no último ano: " << avg_last_year << std::endl;

    //* Maior e Menor Preço nos Últimos 10 anos
    double max_last_10_years = thrust::reduce(d_vec.begin(), d_vec.end(), -1.0, thrust::maximum<double>());
    double min_last_10_years = thrust::reduce(d_vec.begin(), d_vec.end(), 1000000, thrust::minimum<double>());

    std::cout << "Maior preço nos últimos 10 anos: " << max_last_10_years << std::endl;
    std::cout << "Menor preço nos últimos 10 anos: " << min_last_10_years << std::endl;

    //* Maior e Menor Preço no Último ano
    double max_last_year = thrust::reduce(d_vec.end() - 365, d_vec.end(), -1.0, thrust::maximum<double>());
    double min_last_year = thrust::reduce(d_vec.end() - 365, d_vec.end(), 1000000, thrust::minimum<double>());

    std::cout << "Maior preço no último ano: " << max_last_year << std::endl;
    std::cout << "Menor preço no último ano: " << min_last_year << std::endl;

    return 0;
}