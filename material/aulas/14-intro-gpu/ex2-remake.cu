#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

int main() {
    thrust::host_vector<double> h_APPL;
    thrust::host_vector<double> h_MCSF;

    double v1, v2;

    while (cin >> v1 >> v2) {
        h_APPL.push_back(v1);
        h_MCSF.push_back(v2);
    }

    thrust::device_vector<double> d_APPL(h_APPL);
    thrust::device_vector<double> d_MCSF(h_MCSF);

    thrust::device_vector<double> d_diff(d_APPL.size());

    thrust::transform(d_APPL.begin(), d_APPL.end(), d_MCSF.begin(), d_diff.begin(), thrust::minus<double>());

    double mean_diff = thrust::reduce(d_diff.begin(), d_diff.end(), 0.0, thrust::plus<double>()) / d_diff.size();

    cout << "Mean difference: " << mean_diff  << endl;

    //! calcular o desvio padrão das difrenças

    thrust::device_vector<double> mean_diff_vector(d_diff.size(), mean_diff);

    // device_vector (x - u)
    thrust::device_vector<double> d_diff_to_mean(d_diff.size(), 0.0);

    thrust::transform(d_diff.begin(), d_diff.end(),
                     mean_diff_vector.begin(),
                     d_diff_to_mean.begin(),
                     thrust::minus<double>());

    thrust::host_vector<double> h_diff_to_mean(d_diff_to_mean);

    thrust::device_vector<double> d_diff_to_mean2(d_diff_to_mean);
    // (x-u)²
    thrust::transform(d_diff_to_mean.begin(), d_diff_to_mean.end(),
                    d_diff_to_mean2.begin(),
                    thrust::square<double>());

    // somatório (x-u)²
    double sum_square_dif = thrust::reduce(d_diff_to_mean2.begin(), d_diff_to_mean2.end(), 0.0, thrust::plus<float>()) / d_diff_to_mean2.size();

    double std_dev = pow(sum_square_dif, 0.5) ;

    cout << "Desvio Padrão: " << std_dev << endl;


    return 0;
}